#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"
#include "kernels.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb-master/stb_image_write.h"

using namespace std;

int main(int argc, char* argv[]) {

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	if (argc == 1) {
		printf("Please supply the input images.");
		return 0;
	}

	int img_count = argc - 1;
	int width, height, bpp;
	uint8_t* rgb_images[img_count]; //that many images are given as input.
	uint8_t* shifted_rgb_images[img_count];
	uint8_t* d_rgb_images[img_count];
	float* gray_image[PYRAMID_LEVEL * img_count];
	uint8_t* mtb[PYRAMID_LEVEL * img_count];
	uint8_t* ebm[PYRAMID_LEVEL * img_count];
	uint8_t* shifted_mtb[PYRAMID_LEVEL * img_count];
	uint8_t* shifted_ebm[PYRAMID_LEVEL * img_count];


	//hipMalloc((void **) &d_images, sizeof(GPUImage)*PYRAMID_LEVEL*img_count);

	for (int i = 1; i <= img_count; ++i) {
		if (!read_Img(argv[i], rgb_images[i - 1], &width, &height, &bpp)) {
			printf("Could not read image.");
			return 0;
		}

		int nImageSize = width * height; //total pixel count of the image.
		size_t sizeOfImage = nImageSize * sizeof(uint8_t); //size of source image where each pixel is converted to uint8_t.

		hipMalloc((void **) &d_rgb_images[i-1], sizeOfImage*3);
		hipMemcpy(d_rgb_images[i-1], rgb_images[i-1], sizeOfImage*3, hipMemcpyHostToDevice);

		int tmpSizeOfImage = sizeOfImage;
		int tmpWidth = width;
		int tmpHeight = height;
		int tmpNImageSize = nImageSize;

//		texRef.normalized = false;
//		texRef.filterMode = hipFilterModeLinear;
//		size_t pitch[30], tex_ofs[30];
		for(int j = 0; j < PYRAMID_LEVEL; j++, tmpSizeOfImage/=4, tmpWidth/=2, tmpHeight/=2, tmpNImageSize/=4){

//			texRef.normalized = false;
//			texRef.filterMode = hipFilterModeLinear;

			//hipMallocPitch((void**)&(gray_image[(i-1) * PYRAMID_LEVEL + j]), &(pitch[(i-1) * PYRAMID_LEVEL + j]), tmpWidth, tmpHeight);

			// Allocate CUDA array in device memory
//			hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
//						hipChannelFormatKindFloat);
//			hipMallocArray(&(gray_image[(i-1) * PYRAMID_LEVEL + j]), &channelDesc, tmpWidth, tmpHeight);
			hipMalloc((void **) &(gray_image[(i-1) * PYRAMID_LEVEL + j]), tmpNImageSize*sizeof(float));
			hipMalloc((void **) &(mtb[(i-1) * PYRAMID_LEVEL + j]), tmpSizeOfImage);
			hipMalloc((void **) &(ebm[(i-1) * PYRAMID_LEVEL + j]), tmpSizeOfImage);
			hipMalloc((void **) &(shifted_mtb[(i-1) * PYRAMID_LEVEL + j]), tmpSizeOfImage);
			hipMalloc((void **) &(shifted_ebm[(i-1) * PYRAMID_LEVEL + j]), tmpSizeOfImage);

			dim3 dimGrid, dimBlock;
			dimBlock=dim3(16, 16);
			dimGrid=dim3((tmpWidth + dimBlock.x - 1) / dimBlock.x,
						(tmpHeight + dimBlock.y - 1) / dimBlock.y);

			if(j==0){

				convert2_GrayScale<<< dimGrid, dimBlock >>>(gray_image[(i-1) * PYRAMID_LEVEL + j], d_rgb_images[i-1], tmpNImageSize, tmpWidth);
			} else {
//				hipBindTexture2D (&tex_ofs, &texRef, gray_image[(i-1) * PYRAMID_LEVEL + j-1], &texRef.channelDesc, tmpWidth*2, tmpHeight*2, pitch);
//				texRef = ref[(i-1) * PYRAMID_LEVEL + j];

				hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
						hipChannelFormatKindFloat);
				hipArray* cuArray;
				hipMallocArray(&cuArray, &channelDesc, tmpWidth*2, tmpHeight*2);

				// Copy to device memory some data located at address h_data
				// in host memory
				hipMemcpyToArray(cuArray, 0, 0, gray_image[(i-1) * PYRAMID_LEVEL + j -1], tmpNImageSize*sizeof(float)*4, hipMemcpyDeviceToDevice);


				// Specify texture
				struct hipResourceDesc resDesc;
				memset(&resDesc, 0, sizeof(resDesc));
				resDesc.resType = hipResourceTypeArray;
				resDesc.res.array.array = cuArray;

				// Specify texture object parameters
				struct hipTextureDesc texDesc;
				memset(&texDesc, 0, sizeof(texDesc));
				texDesc.addressMode[0] = hipAddressModeClamp;
				texDesc.addressMode[1] = hipAddressModeClamp;
				texDesc.filterMode = hipFilterModeLinear;
				texDesc.readMode = hipReadModeElementType;
				texDesc.normalizedCoords = 1;

				// Create texture object
				hipTextureObject_t texObj = 0;
				hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

				// Invoke kernel
				dim3 dimBlock(16, 16);
				dim3 dimGrid((tmpWidth + dimBlock.x - 1) / dimBlock.x,
						(tmpHeight + dimBlock.y - 1) / dimBlock.y);

				transformKernel<<<dimGrid, dimBlock>>>(gray_image[(i-1) * PYRAMID_LEVEL + j], texObj, tmpWidth, tmpHeight);

				hipDestroyTextureObject(texObj);

				// Free device memory
				hipFreeArray(cuArray);
//				downsample<<<dimGrid, dimBlock>>>(gray_image[(i-1) * PYRAMID_LEVEL + j], tmpWidth, tmpHeight);
//				hipDeviceSynchronize();
//				hipUnbindTexture(texRef);gray_image
//				hipFree(gray_image[(i-1) * PYRAMID_LEVEL + j-1]);
			}

//TODO dynamic parallelism kullanamadik kernel icinde kernel. cok kasti. hipMalloc vs dinamik size'da yapamadik kernel icinde.
//TODO texture cok vaktimizi aldi. (2 gün totalde) globalden texRef alan yontem imaj piramidinin bazi levellarinda calismadi,
//TODO benchmark using 2 kernels of histogram finding, and 1 merged kernel.
//TODO findMedian kernel'ini de gom ustteki merge haline.

//			char str[12];
//			sprintf(str, "%d.png", (i-1) * PYRAMID_LEVEL + j);
//			char path[80]="/home/kca/Desktop/test_mtb";
//			strcat(path, str);
//
//			float* tmpmtb = (float *)malloc(sizeof(float)*tmpNImageSize);
//			hipMemcpy(tmpmtb, gray_image[(i-1) * PYRAMID_LEVEL + j], sizeof(float)*tmpNImageSize, hipMemcpyDeviceToHost);
//			uint8_t* tmpmtbuint;
//			tmpmtbuint = (uint8_t*)malloc(tmpNImageSize);
//			for (int var = 0; var < tmpNImageSize; ++var) {
//				tmpmtbuint[var] = tmpmtb[var];
//			}
//			stbi_write_png(path, tmpWidth, tmpHeight, 1, tmpmtbuint, tmpWidth);
//
			dimBlock=dim3(BLOCK_SIZE);
			dimGrid=dim3(32);

			int* hist;
			hipMalloc((void **)&hist, BLOCK_SIZE * sizeof(int) * 32);

			histogram_smem_atomics<<<dimGrid, dimBlock>>>(gray_image[(i-1) * PYRAMID_LEVEL + j], hist, tmpNImageSize);
			histogram_final_accum<<<1, 256>>>(BLOCK_SIZE*dimGrid.x, hist);

			int* median; hipMalloc((void **)&median, sizeof(int));
			find_Median<<<1, 1>>>(tmpNImageSize, hist, median);

			dimBlock=dim3(16, 16);
			dimGrid=dim3((tmpWidth + dimBlock.x - 1) / dimBlock.x,
						(tmpHeight + dimBlock.y - 1) / dimBlock.y);

			find_Mtb_Ebm<<<dimGrid, dimBlock>>>(gray_image[(i-1) * PYRAMID_LEVEL + j], median, mtb[(i-1) * PYRAMID_LEVEL + j],
					ebm[(i-1) * PYRAMID_LEVEL + j], tmpHeight, tmpWidth);

//			uint8_t* tmpmtb = (uint8_t *)malloc(sizeof(uint8_t)*tmpNImageSize);
//			hipMemcpy(tmpmtb, mtb[(i-1) * PYRAMID_LEVEL + j], sizeof(uint8_t)*tmpNImageSize, hipMemcpyDeviceToHost);
//			stbi_write_png("/home/kca/Desktop/test_mtb.png", tmpWidth, tmpHeight, 1, tmpmtb, tmpWidth);
//			int c = 0;

//			int h_median;
//			hipMemcpy(&h_median, median, sizeof(int), hipMemcpyDeviceToHost);

			hipFree(hist);
			hipFree(median);

//			hipBindTexture2D (&(tex_ofs[(i-1) * PYRAMID_LEVEL + j]), &texRef, gray_image[(i-1) * PYRAMID_LEVEL + j], &texRef.channelDesc,
//					tmpWidth, tmpHeight, pitch[(i-1) * PYRAMID_LEVEL + j]);


		}

//		uint8_t* shifted;
//		hipMalloc((void **)&shifted, width * height *  sizeof(uint8_t));
//
//	    hipMemset(shifted, 255, width * height *  sizeof(uint8_t));
//
//	    int x_shift=-200, y_shift=200;
//
//	    int j_x, i_y, j_width, i_height;
//
//	    if(y_shift < 0) { //height i
//			i_y = -y_shift;
//			i_height = height;
//		}
//		else {
//			i_y = 0;
//			i_height = height - y_shift;
//		}
//
//		if(x_shift < 0) {//width j
//			j_x = -x_shift;
//			j_width = width;
//		}
//		else {
//			j_x = 0;
//			j_width = width - x_shift;
//		}
//
//
//
//
//		dimBlock=dim3(16, 16);
//		dimGrid=dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
//					((i_height) + dimBlock.y - 1) / dimBlock.y);
//
//		shift_Image<<<dimGrid, dimBlock>>>(shifted, d_images_grayscale[i-1], width, height, x_shift, y_shift, j_x, i_y, j_width , i_height);
	}



//**********************************************************************************************************************************
    std::vector<shift_pair> all_shifts;

    int mid_img_index = img_count / 2 + 1;

    for (int m = mid_img_index - 1; m >= 0; --m) {
        all_shifts.emplace_back(calculateOffset(m+1, m, width, height,gray_image, mtb, ebm, shifted_mtb, shifted_ebm));
    }

    //cout << " ilk parttaki imajlari shiftliyoruz tek tek ..." << endl;

    int k = 0, eskiTotalX = 0, eskiTotalY = 0;
    for (int m = mid_img_index - 1; m >= 0; --m) {

//    	all_images[m].finalShift(all_shifts[k].x + eskiTotalX, all_shifts[k].y + eskiTotalY);

		char str[12];
		sprintf(str, "m%d-k%d.png", m,k);
		char path[80]="/home/berkay/Desktop/out/shiftedGray";
		strcat(path, str);



		int x_shift=all_shifts[k].x + eskiTotalX, y_shift=all_shifts[k].y + eskiTotalY; //TODO check those

		int tmpWidth = width;
		int tmpHeight = height;
		int tmpNImageSize = tmpWidth * tmpHeight;

		hipMalloc((void**)&shifted_rgb_images[m], 3 * tmpNImageSize);
		hipMemset(shifted_rgb_images[m], 0, 3*tmpNImageSize * sizeof(uint8_t));

		int j_x, i_y, j_width, i_height;

		if(y_shift < 0) { //height i
			i_y = -y_shift;
			i_height = tmpHeight;
		}
		else {
			i_y = 0;
			i_height = tmpHeight - y_shift;
		}

		if(x_shift < 0) {//width j
			j_x = -x_shift;
			j_width = tmpWidth;
		}
		else {
			j_x = 0;
			j_width = tmpWidth - x_shift;
		}

		dim3 dimBlock=dim3(16, 16);
		dim3 dimGrid=dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
					((i_height) + dimBlock.y - 1) / dimBlock.y);

//		uint8_t* tmpShiftedGray;
//		hipMalloc((void **) &tmpShiftedGray,tmpNImageSize);
//		hipMemset(tmpShiftedGray, 0, tmpNImageSize * sizeof(uint8_t));

		RGB_shift_Image<<<dimGrid, dimBlock>>>(shifted_rgb_images[m], d_rgb_images[m], tmpWidth, tmpHeight, x_shift, y_shift, j_x, i_y, j_width , i_height);

		uint8_t* tmpmtb = (uint8_t *)malloc(sizeof(uint8_t)*tmpNImageSize*3);
		hipMemcpy(tmpmtb, shifted_rgb_images[m], sizeof(uint8_t)*tmpNImageSize*3, hipMemcpyDeviceToHost);

		stbi_write_png(path, tmpWidth, tmpHeight, 3, tmpmtb, tmpWidth*3);

        eskiTotalX += all_shifts[k].x;
        eskiTotalY += all_shifts[k].y;
        k++;
        //cout << "   shiftledik: x,y " << eskiTotalX << " " << eskiTotalY << endl;
    }

    //cout << "ikinci part baslar ..." << endl;
    all_shifts.clear();

    for (int m = mid_img_index + 1; m < img_count; ++m) {
        all_shifts.emplace_back(calculateOffset(m-1, m, width, height, gray_image,mtb, ebm, shifted_mtb, shifted_ebm));
    }

    //cout << " ikinci parttaki imajlari shiftliyoruz tek tek ..." << endl;

    k = 0;
    eskiTotalX = 0;
    eskiTotalY = 0;
    for (int m = mid_img_index + 1; m < img_count; ++m) {
        //all_images[m].finalShift(all_shifts[k].x + eskiTotalX, all_shifts[k].y + eskiTotalY);



    	char str[12];
		sprintf(str, "m%d-k%d.png", m,k);
		char path[80]="/home/berkay/Desktop/out/shiftedGray";
		strcat(path, str);



		int x_shift=all_shifts[k].x + eskiTotalX, y_shift=all_shifts[k].y + eskiTotalY; //TODO check those

		int tmpWidth = width;
		int tmpHeight = height;
		int tmpNImageSize = tmpWidth * tmpHeight;

		hipMalloc((void**)&shifted_rgb_images[m], 3 * tmpNImageSize);
		hipMemset(shifted_rgb_images[m], 0, 3*tmpNImageSize * sizeof(uint8_t));

		int j_x, i_y, j_width, i_height;

		if(y_shift < 0) { //height i
			i_y = -y_shift;
			i_height = tmpHeight;
		}
		else {
			i_y = 0;
			i_height = tmpHeight - y_shift;
		}

		if(x_shift < 0) {//width j
			j_x = -x_shift;
			j_width = tmpWidth;
		}
		else {
			j_x = 0;
			j_width = tmpWidth - x_shift;
		}

		dim3 dimBlock=dim3(16, 16);
		dim3 dimGrid=dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
					((i_height) + dimBlock.y - 1) / dimBlock.y);

//		uint8_t* tmpShiftedGray;
//		hipMalloc((void **) &tmpShiftedGray,tmpNImageSize);
//		hipMemset(tmpShiftedGray, 0, tmpNImageSize * sizeof(uint8_t));

		RGB_shift_Image<<<dimGrid, dimBlock>>>(shifted_rgb_images[m], d_rgb_images[m], tmpWidth, tmpHeight, x_shift, y_shift, j_x, i_y, j_width , i_height);

		uint8_t* tmpmtb = (uint8_t *)malloc(sizeof(uint8_t)*tmpNImageSize*3);
		hipMemcpy(tmpmtb, shifted_rgb_images[m], sizeof(uint8_t)*tmpNImageSize*3, hipMemcpyDeviceToHost);

		stbi_write_png(path, tmpWidth, tmpHeight, 3, tmpmtb, tmpWidth*3);



        eskiTotalX += all_shifts[k].x;
        eskiTotalY += all_shifts[k].y;
        k++;
        //cout << "   shiftledik: x,y " << eskiTotalX << " " << eskiTotalY << endl;
    }



    //print original grayscale img

	char str[12];
	sprintf(str, ".png");
	char path[80]="/home/berkay/Desktop/out/orgi";
	strcat(path, str);

	stbi_write_png(path, width, height, 3, rgb_images[mid_img_index], width*3);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<milliseconds<<endl;




	//printf("Done..........\n");

	//stbi_write_png("/home/kca/Desktop/test1.png", width, height, 1, gray_images[0], width);
	//stbi_write_png("/home/kca/Desktop/test2.png", width, height, 1, gray_images[1], width);
	//stbi_write_png("/home/kca/Desktop/test3.png", width, height, 1, gray_images[2], width);
	//stbi_write_png("/home/kca/Desktop/test4.png", width, height, 1, gray_images[3], width);
	//stbi_write_png("/home/kca/Desktop/test5.png", width, height, 1, gray_images[4], width);

	return 0;
}
