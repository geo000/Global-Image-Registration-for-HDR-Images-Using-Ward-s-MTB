#include "hip/hip_runtime.h"
// Implementation is done by:
// Kadir Cenk Alpay
// Kadir Berkay Aydemir
// until the rotation part. Rotation part is done only by:
// Kadir Cenk Alpay

// Rotation brute force tested: -1.0f <= x <= 1.0f, by 0.2 degree difference.

//TODO dynamic parallelism kullanamadik kernel icinde kernel. cok kasti. hipMalloc vs dinamik size'da yapamadik kernel icinde.
//TODO texture cok vaktimizi aldi. (2 gün totalde) globalden texRef alan yontem imaj piramidinin bazi levellarinda calismadi,
//TODO benchmark using 2 kernels of histogram finding, and 1 merged kernel.
//TODO findMedian kernel'ini de gom ustteki merge haline.
//TODO write'ler olmadan, readler dahil 1080 -> 593.515ms. gtx850m -> 1638.57 ms.
//TODO preprocessing kisminda img_count tane stream actik, her mtb ebm find kernel'i kendi streaminde. memcpyAsync. gtx850m->1495.27 ms oldu.
//TODO asil algoritmanin kostugu kismi da her imajin kendi stream'ine koyduk. ama calculateOffsetError if checki sequential hala. simdi multithreaded ekleyecegiz.
//TODO asil algoritmaya multithread ekledik (image_count tane thread her foto 1 cekirdekte isleniyor) ve -O3 optimizasyon actik (compiler flag), gtx850m -> 1079.43 ms
//TODO Release modda cesitli compiler optimizasyonlari eklendi:
//TODO 4k -> (4032x2268) ve 2k -> 1536x2048 dikey.
//TODO ustteki satir icin: gtx850m -> 343.15 ms (2k fotolar 5 tane), 933.248 ms (4k fotolar, 5 tane).
//TODO aynisini CPU (fully optimized -O3 var, ama single threaded)-> 1237.07ms (2k fotolar 5 tane) 3885.8ms (4k fotolar 5 tane)
//TODO CPU multithread yazdik alignment algo. kismina, aksine yavaslatti 4011.04 ms oldu 4k 5 foto. 1401.87 ms 2k 5 foto. (gtx850m)
//TODO #define THREAD_COUNT 16 yapmistik tum yukaridaki sonuclari. 32 ve 8 denedik, sure degismedi.

#include <iostream>
#include <vector>
#include <pthread.h>
#include "hip/hip_runtime.h"
#include "kernels.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb-master/stb_image_write.h"

using namespace std;

int ANGLE_STEP_TESTED = 11;

bool readImageFromFile(char *filename, uint8_t*& img, int* width, int* height,
		int* bpp) {

	img = stbi_load(filename, width, height, bpp, 3);

	if (img) {
		return true;
	} else {
		return false;
	}
}
//
//void* calculateOffset(void * args) {
//
//	arguments *_arg;
//	_arg = (arguments *) args;
//
//	int width = _arg->width;
//	int height = _arg->height;
//	int first_index = _arg->first_ind;
//	int second_index = _arg->second_ind;
//	dim3 dimGrid, dimBlock;
//	int tmpWidth = width / (pow(2, PYRAMID_LEVEL - 1));
//	int tmpHeight = height / (pow(2, PYRAMID_LEVEL - 1));
//	int tmpNImageSize = tmpWidth * tmpHeight;
//
//	uint8_t** mtb = _arg->mtb;
//	uint8_t** ebm = _arg->ebm;
//	uint8_t** shifted_mtb = _arg->shifted_mtb;
//	uint8_t** shifted_ebm = _arg->shifted_ebm;
//	hipStream_t stream = _arg->stream;
//
//	int curr_level = PYRAMID_LEVEL - 1;
//	int curr_offset_x = 0;
//	int curr_offset_y = 0;
//	int offset_return_x = 0;
//	int offset_return_y = 0;
//
//	for (int k = curr_level; k >= 0;
//			--k, tmpWidth *= 2, tmpHeight *= 2, tmpNImageSize *= 4) {
//		curr_offset_x = 2 * offset_return_x;
//		curr_offset_y = 2 * offset_return_y;
//
//		int min_error = 255 * height * width;
//
//		for (int i = -1; i <= 1; ++i) {
//			for (int j = -1; j <= 1; ++j) {
//				int xs = curr_offset_x + i;
//				int ys = curr_offset_y + j;
//
//				int x_shift = xs, y_shift = ys; //TODO check those
//
//				int j_x, i_y, j_width, i_height;
//
//				if (y_shift < 0) { //height i
//					i_y = -y_shift;
//					i_height = tmpHeight;
//				} else {
//					i_y = 0;
//					i_height = tmpHeight - y_shift;
//				}
//
//				if (x_shift < 0) { //width j
//					j_x = -x_shift;
//					j_width = tmpWidth;
//				} else {
//					j_x = 0;
//					j_width = tmpWidth - x_shift;
//				}
//
//				dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//				dimGrid = dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
//						((i_height) + dimBlock.y - 1) / dimBlock.y);
//
//				hipMemset(shifted_mtb[second_index * PYRAMID_LEVEL + k], 0,
//						tmpNImageSize * sizeof(uint8_t));
//				hipMemset(shifted_ebm[second_index * PYRAMID_LEVEL + k], 0,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *xor_result;
//				hipMalloc((void **) &xor_result,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *after_first_and;
//				hipMalloc((void **) &after_first_and,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *after_second_and;
//				hipMalloc((void **) &after_second_and,
//						tmpNImageSize * sizeof(uint8_t));
//
//				int* err;
//				int error;
//				hipMalloc((void **) &err, sizeof(int));
//
//				shift_Image<<<dimGrid, dimBlock, 0, stream>>>(
//						shifted_mtb[second_index * PYRAMID_LEVEL + k],
//						mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
//				shift_Image<<<dimGrid, dimBlock, 0, stream>>>(
//						shifted_ebm[second_index * PYRAMID_LEVEL + k],
//						ebm[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
//
//				dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//				dimGrid = dim3(((tmpWidth) + dimBlock.x - 1) / dimBlock.x,
//						((tmpHeight) + dimBlock.y - 1) / dimBlock.y);
//
//				XOR<<<dimGrid, dimBlock, 0, stream>>>(xor_result,
//						mtb[first_index * PYRAMID_LEVEL + k],
//						shifted_mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpNImageSize);
//
//				AND<<<dimGrid, dimBlock, 0, stream>>>(after_first_and,
//						ebm[first_index * PYRAMID_LEVEL + k], xor_result,
//						tmpWidth, tmpNImageSize);
//
//				AND<<<dimGrid, dimBlock, 0, stream>>>(after_second_and,
//						shifted_ebm[second_index * PYRAMID_LEVEL + k],
//						after_first_and, tmpWidth, tmpNImageSize);
//
//				count_Errors<<<32, 256, 0, stream>>>(after_second_and, err,
//						tmpNImageSize);
//
//				hipMemcpy(&error, err, sizeof(int), hipMemcpyDeviceToHost); //CANNOT BE ASYNC
//
//				if (error < min_error) {
//					offset_return_x = xs;
//					offset_return_y = ys;
//					min_error = error;
//				}
//				//hipFree(err);
//			}
//		}
//	}
//
//	_arg->shiftp->x = curr_offset_x;
//	_arg->shiftp->y = curr_offset_y;
//
//	cout << "found offset (x, y): " << curr_offset_x << " " << curr_offset_y
//			<< endl;
//
//	pthread_exit(NULL);
//}

int main(int argc, char* argv[]) {
	int width, height, bpp;
	int img_count = argc - 1;

	if (argc == 1) {
		printf("Please supply the input images.");
		return 0;
	}

	uint8_t* input_rgb_images[img_count]; //that many images are given as input.
	uint8_t* shifted_rgb_images[img_count];
	uint8_t* device_rgb_images[img_count];
	uint8_t* rotated_rgb_images[ANGLE_STEP_TESTED];
	uint8_t* ready_rgb_images[ANGLE_STEP_TESTED + 1];

	float* gray_image[PYRAMID_LEVEL * (ANGLE_STEP_TESTED + 1)];
	uint8_t* mtb[PYRAMID_LEVEL * (ANGLE_STEP_TESTED + 1)];
	uint8_t* ebm[PYRAMID_LEVEL * (ANGLE_STEP_TESTED + 1)];
	uint8_t* shifted_mtb[PYRAMID_LEVEL * (ANGLE_STEP_TESTED + 1)];
	uint8_t* shifted_ebm[PYRAMID_LEVEL * (ANGLE_STEP_TESTED + 1)];

	int imgPixelCount = width * height; //total pixel count of the image.
	size_t imgByteSize = imgPixelCount * sizeof(uint8_t); //size of source image where each pixel is converted to uint8_t.
	size_t imgByteSizeRgb = imgByteSize * 3;

	//Read the input images
	for (int i = 1; i <= img_count; ++i) {
		//all input images are of the same width and height.
		if (!readImageFromFile(argv[i], input_rgb_images[i - 1], &width,
				&height, &bpp)) {
			printf("Could not read image.");
			return 0;
		}

		imgPixelCount = width * height; //total pixel count of the image.
		imgByteSize = imgPixelCount * sizeof(uint8_t); //size of source image where each pixel is converted to uint8_t.
		imgByteSizeRgb = imgByteSize * 3;

		//Send the images to the device memory
		hipMalloc((void **) &device_rgb_images[i - 1], imgByteSizeRgb);
		hipMemcpy(device_rgb_images[i - 1], input_rgb_images[i - 1],
				imgByteSizeRgb, hipMemcpyHostToDevice);
	}

	//Kernel preparation
	dim3 dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
	dim3 dimGrid = dim3((width + dimBlock.x - 1) / dimBlock.x,
			(height + dimBlock.y - 1) / dimBlock.y);

	//Get the rotated versions of the second image ready
	for (int var = 0; var < ANGLE_STEP_TESTED; ++var) {
		float step = 0.2f;
		float angle = ((-1.0f + (var * step)) * M_PI) / 180.0f;

		hipMalloc((void **) &(rotated_rgb_images[var]), imgByteSizeRgb);
		hipMemset(rotated_rgb_images[var], 0, imgByteSizeRgb);

		//Rotate the second image
		rotate<<<dimGrid, dimBlock>>>(rotated_rgb_images[var],
				device_rgb_images[1], width, height, angle);

//		hipDeviceSynchronize();
//
//		char str[12];
//		sprintf(str, "%d.png", var);
//		char path[80] = "/home/kca/Desktop/img";
//		strcat(path, str);
//
//		uint8_t* tmpmtb = (uint8_t *) malloc(imgByteSizeRgb);
//		hipMemcpy(tmpmtb, rotated_rgb_images[var], imgByteSizeRgb,
//				hipMemcpyDeviceToHost);
//
//		stbi_write_png(path, width, height, 3, tmpmtb, width * 3);
	}

	hipDeviceSynchronize();

	//all images are ready
	ready_rgb_images[0] = device_rgb_images[0];

	int var = 0;
	char str[12];
	sprintf(str, "%d.png", var);
	char path[80] = "/home/kca/Desktop/img";
	strcat(path, str);

	uint8_t* tmpmtb = (uint8_t *) malloc(imgByteSizeRgb);
	hipMemcpy(tmpmtb, ready_rgb_images[var], imgByteSizeRgb,
			hipMemcpyDeviceToHost);

	stbi_write_png(path, width, height, 3, tmpmtb, width * 3);

	for (int var = 0; var < ANGLE_STEP_TESTED; ++var) {
		//cout<<"var and digeri: "<<var <<" "<<ANGLE_STEP_TESTED+1<<endl;

		ready_rgb_images[var+1] = rotated_rgb_images[var];

		char str[32];
		sprintf(str, "%d.png", var);
		char path[80] = "/home/kca/Desktop/img";
		strcat(path, str);

		uint8_t* tmpmtb = (uint8_t *) malloc(imgByteSizeRgb);
		hipMemcpy(tmpmtb, ready_rgb_images[var+1], imgByteSizeRgb,
				hipMemcpyDeviceToHost);

		stbi_write_png(path, width, height, 3, tmpmtb, width * 3);
	}


	return 0;

	for (int i = 0; i < (ANGLE_STEP_TESTED + 1); ++i) {
		int tmpSizeOfImage = imgByteSize;
		int tmpWidth = width;
		int tmpHeight = height;
		int tmpNImageSize = imgPixelCount;

		for (int j = 0; j < PYRAMID_LEVEL; j++, tmpSizeOfImage /= 4, tmpWidth /=
				2, tmpHeight /= 2, tmpNImageSize /= 4) {

			hipMalloc((void **) &(gray_image[i * PYRAMID_LEVEL + j]),
					tmpNImageSize * sizeof(float));
			hipMalloc((void **) &(mtb[i * PYRAMID_LEVEL + j]), tmpSizeOfImage);
			hipMalloc((void **) &(ebm[i * PYRAMID_LEVEL + j]), tmpSizeOfImage);
			hipMalloc((void **) &(shifted_mtb[i * PYRAMID_LEVEL + j]),
					tmpSizeOfImage);
			hipMalloc((void **) &(shifted_ebm[i * PYRAMID_LEVEL + j]),
					tmpSizeOfImage);

			dim3 dimGrid, dimBlock;
			dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
			dimGrid = dim3((tmpWidth + dimBlock.x - 1) / dimBlock.x,
					(tmpHeight + dimBlock.y - 1) / dimBlock.y);

			if (j == 0) {
				convertToGrayscale<<<dimGrid, dimBlock>>>(
						gray_image[i * PYRAMID_LEVEL + j], ready_rgb_images[i],
						tmpNImageSize, tmpWidth);
				//hipDeviceSynchronize();
			} else {
				//				hipBindTexture2D (&tex_ofs, &texRef, gray_image[(i-1) * PYRAMID_LEVEL + j-1], &texRef.channelDesc, tmpWidth*2, tmpHeight*2, pitch);
				//				texRef = ref[(i-1) * PYRAMID_LEVEL + j];

				hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0,
						0, 0, hipChannelFormatKindFloat);
				hipArray* cuArray;
				hipMallocArray(&cuArray, &channelDesc, tmpWidth * 2,
						tmpHeight * 2);

				// Copy to device memory some data located at address h_data
				// in host memory
				hipMemcpyToArray(cuArray, 0, 0,
						gray_image[i * PYRAMID_LEVEL + j - 1],
						tmpNImageSize * sizeof(float) * 4,
						hipMemcpyDeviceToDevice);

				// Specify texture
				struct hipResourceDesc resDesc;
				memset(&resDesc, 0, sizeof(resDesc));
				resDesc.resType = hipResourceTypeArray;
				resDesc.res.array.array = cuArray;

				// Specify texture object parameters
				struct hipTextureDesc texDesc;
				memset(&texDesc, 0, sizeof(texDesc));
				texDesc.addressMode[0] = hipAddressModeClamp;
				texDesc.addressMode[1] = hipAddressModeClamp;
				texDesc.filterMode = hipFilterModeLinear;
				texDesc.readMode = hipReadModeElementType;
				texDesc.normalizedCoords = 1;

				// Create texture object
				hipTextureObject_t texObj = 0;
				hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

				// Invoke kernel
				dim3 dimBlock(THREAD_COUNT, THREAD_COUNT);
				dim3 dimGrid((tmpWidth + dimBlock.x - 1) / dimBlock.x,
						(tmpHeight + dimBlock.y - 1) / dimBlock.y);

				downsample<<<dimGrid, dimBlock>>>(
						gray_image[i * PYRAMID_LEVEL + j], texObj, tmpWidth,
						tmpHeight);

				//hipDestroyTextureObject(texObj);

				// Free device memory
				//hipFreeArray(cuArray);
				//				downsample<<<dimGrid, dimBlock>>>(gray_image[(i-1) * PYRAMID_LEVEL + j], tmpWidth, tmpHeight);
				//				hipDeviceSynchronize();
				//				hipUnbindTexture(texRef);gray_image
				//				hipFree(gray_image[(i-1) * PYRAMID_LEVEL + j-1]);
			}

			//
			dimBlock = dim3(BLOCK_SIZE);
			dimGrid = dim3(32);

			int* hist;
			hipMalloc((void **) &hist, BLOCK_SIZE * sizeof(int) * 32);

			histogramSmemAtomics<<<dimGrid, dimBlock>>>(
					gray_image[i * PYRAMID_LEVEL + j], hist, tmpNImageSize);
			histogramFinalAccumulate<<<1, 256>>>(
			BLOCK_SIZE * dimGrid.x, hist);

			int* median;
			hipMalloc((void **) &median, sizeof(int));
			findMedian<<<1, 1>>>(tmpNImageSize, hist, median);

			dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
			dimGrid = dim3((tmpWidth + dimBlock.x - 1) / dimBlock.x,
					(tmpHeight + dimBlock.y - 1) / dimBlock.y);

			calculateMtbEbm<<<dimGrid, dimBlock>>>(
					gray_image[i * PYRAMID_LEVEL + j], median,
					mtb[i * PYRAMID_LEVEL + j], ebm[i * PYRAMID_LEVEL + j],
					tmpHeight, tmpWidth);

		}
	}

//	hipDeviceSynchronize(); //wait for all streams to finish up.
//
////**********************************************************************************************************************************
//	shift_pair all_shifts[img_count];
//
//	//test ----------------------------------------
//
////	for (float angle = (-1.0f * M_PI) / 180.0f; angle <= (1.0f * M_PI) / 180.0f;
////			angle += (0.2f * M_PI) / 180.0f) {
////
////		//float angle = (2.0f*M_PI)/180.0f;
////
////		dim3 dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
////		dim3 dimGrid = dim3((width + dimBlock.x - 1) / dimBlock.x,
////				(height + dimBlock.y - 1) / dimBlock.y);
////
////		uint8_t* test_rgb;
////		hipMalloc((void **) &test_rgb, height * width * sizeof(uint8_t) * 3);
////		hipMemset(test_rgb, 0, 3 * height * width * sizeof(uint8_t));
////
//////	hipMemcpyAsync(test_rgb, rgb_images[0], height*width*sizeof(uint8_t) * 3,
//////			hipMemcpyHostToDevice, streams[0]);
////
////		//Rotate(float* Source, float* Destination, int sizeX, int sizeY, float deg)
////		Rotate<<<dimGrid, dimBlock, 0, streams[0]>>>(d_rgb_images[0], test_rgb,
////				width, height, angle);
////
////		hipDeviceSynchronize();
////
////		char str[32];
////		if (angle > 0)
////			sprintf(str, "%d%f.png", 0, angle);
////		else
////			sprintf(str, "%d%f%d.png", 0, -1.0f * angle, 77);
////		char path[80] = "/home/kca/Desktop/rotated";
////		strcat(path, str);
////
////		uint8_t* tmpmtb = (uint8_t *) malloc(
////				sizeof(uint8_t) * height * width * sizeof(uint8_t) * 3);
////		hipMemcpy(tmpmtb, test_rgb,
////				sizeof(uint8_t) * height * width * sizeof(uint8_t) * 3,
////				hipMemcpyDeviceToHost);
////
////		stbi_write_png(path, width, height, 3, tmpmtb, width * 3);
////
////		hipFree(test_rgb);
////
////	}
////
////	return 0;
//
//	//---------------------------------------------
//
//	int mid_img_index = img_count / 2;
//
////	for (int m = mid_img_index - 1; m >= 0; --m) {
////		arguments* args = (arguments *) malloc(sizeof(arguments));
////		args->shiftp = &(all_shifts[m]);
////		args->stream = streams[m];
////		args->first_ind = m + 1;
////		args->second_ind = m;
////		args->width = width;
////		args->height = height;
////		args->mtb = mtb;
////		args->ebm = ebm;
////		args->shifted_mtb = shifted_mtb;
////		args->shifted_ebm = shifted_ebm;
////
////		//cout << "part1 thread create ediyoz..." << endl;
////		int rc = pthread_create(&threads[m], NULL, calculateOffset,
////				(void *) args);
////
////		if (rc) {
////			cout << "Error:unable to create thread," << rc << endl;
////			exit(-1);
////		}
//
//	int curr_level = PYRAMID_LEVEL - 1;
//	int curr_offset_x = 0;
//	int curr_offset_y = 0;
//	int offset_return_x = 0;
//	int offset_return_y = 0;
//	int first_index = 0;
//	int second_index = 1;
//	int tmpWidth = width / (pow(2, PYRAMID_LEVEL - 1));
//	int tmpHeight = height / (pow(2, PYRAMID_LEVEL - 1));
//	int tmpNImageSize = tmpWidth * tmpHeight;
//
//	int last_error = 9999;
//
//	for (int k = curr_level; k >= 0;
//			--k, tmpWidth *= 2, tmpHeight *= 2, tmpNImageSize *= 4) {
//		curr_offset_x = 2 * offset_return_x;
//		curr_offset_y = 2 * offset_return_y;
//
//		int min_error = 255 * height * width;
//
//		for (int i = -1; i <= 1; ++i) {
//			for (int j = -1; j <= 1; ++j) {
//				int xs = curr_offset_x + i;
//				int ys = curr_offset_y + j;
//
//				int x_shift = xs, y_shift = ys;
//
//				int j_x, i_y, j_width, i_height;
//
//				if (y_shift < 0) { //height i
//					i_y = -y_shift;
//					i_height = tmpHeight;
//				} else {
//					i_y = 0;
//					i_height = tmpHeight - y_shift;
//				}
//
//				if (x_shift < 0) { //width j
//					j_x = -x_shift;
//					j_width = tmpWidth;
//				} else {
//					j_x = 0;
//					j_width = tmpWidth - x_shift;
//				}
//
//				dim3 dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//				dim3 dimGrid = dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
//						((i_height) + dimBlock.y - 1) / dimBlock.y);
//
//				hipMemset(shifted_mtb[second_index * PYRAMID_LEVEL + k], 0,
//						tmpNImageSize * sizeof(uint8_t));
//				hipMemset(shifted_ebm[second_index * PYRAMID_LEVEL + k], 0,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *xor_result;
//				hipMalloc((void **) &xor_result,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *after_first_and;
//				hipMalloc((void **) &after_first_and,
//						tmpNImageSize * sizeof(uint8_t));
//
//				uint8_t *after_second_and;
//				hipMalloc((void **) &after_second_and,
//						tmpNImageSize * sizeof(uint8_t));
//
//				int* err;
//				int error;
//				hipMalloc((void **) &err, sizeof(int));
//
//				shift_Image<<<dimGrid, dimBlock>>>(
//						shifted_mtb[second_index * PYRAMID_LEVEL + k],
//						mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
//				shift_Image<<<dimGrid, dimBlock>>>(
//						shifted_ebm[second_index * PYRAMID_LEVEL + k],
//						ebm[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
//
//				dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//				dimGrid = dim3(((tmpWidth) + dimBlock.x - 1) / dimBlock.x,
//						((tmpHeight) + dimBlock.y - 1) / dimBlock.y);
//
//				XOR<<<dimGrid, dimBlock>>>(xor_result,
//						mtb[first_index * PYRAMID_LEVEL + k],
//						shifted_mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
//						tmpNImageSize);
//
//				AND<<<dimGrid, dimBlock>>>(after_first_and,
//						ebm[first_index * PYRAMID_LEVEL + k], xor_result,
//						tmpWidth, tmpNImageSize);
//
//				AND<<<dimGrid, dimBlock>>>(after_second_and,
//						shifted_ebm[second_index * PYRAMID_LEVEL + k],
//						after_first_and, tmpWidth, tmpNImageSize);
//
//				count_Errors<<<32, 256>>>(after_second_and, err, tmpNImageSize);
//
//				hipMemcpy(&error, err, sizeof(int), hipMemcpyDeviceToHost); //CANNOT BE ASYNC
//
//				if (error < min_error) {
//					offset_return_x = xs;
//					offset_return_y = ys;
//					min_error = error;
//				}
//				//hipFree(err);
//			}
//		}
//		last_error = min_error;
//		cout << "last_error: " << last_error << endl;
//	}
//
//	if (last_error < global_min_error) {
//		global_min_error = last_error;
//		global_offset_return_x = offset_return_x;
//		global_offset_return_y = offset_return_y;
//		global_min_error_angle = angle;
//		angle += (0.2f * M_PI) / 180.0f;
//
//		if (!(angle > (1.0f * M_PI) / 180.0f)) {
//			goto START;
//		} else {
//			angle -= (0.2f * M_PI) / 180.0f;
//		}
//	}
//
//	cout << "found offset and angle (x, y, angle): " << global_offset_return_x
//			<< " " << global_offset_return_y << " " << global_min_error_angle
//			<< endl;
//
////	}
////	for (int m = mid_img_index + 1; m < img_count; ++m) {
////
////		arguments* args = (arguments *) malloc(sizeof(arguments));
////		args->shiftp = &(all_shifts[m]);
////		args->stream = streams[m];
////		args->first_ind = m - 1;
////		args->second_ind = m;
////		args->width = width;
////		args->height = height;
////		args->mtb = mtb;
////		args->ebm = ebm;
////		args->shifted_mtb = shifted_mtb;
////		args->shifted_ebm = shifted_ebm;
////
////		//cout << "part2 thread create ediyoz..." << endl;
////		int rc = pthread_create(&threads[m], NULL, calculateOffset,
////				((void *) args));
////
////		if (rc) {
////			cout << "Error:unable to create thread," << rc << endl;
////			exit(-1);
////		}
////
////		int curr_level = PYRAMID_LEVEL - 1;
////		int curr_offset_x = 0;
////		int curr_offset_y = 0;
////		int offset_return_x = 0;
////		int offset_return_y = 0;
////
////		for (int k = curr_level; k >= 0;
////				--k, tmpWidth *= 2, tmpHeight *= 2, tmpNImageSize *= 4) {
////			curr_offset_x = 2 * offset_return_x;
////			curr_offset_y = 2 * offset_return_y;
////
////			int min_error = 255 * height * width;
////
////			for (int i = -1; i <= 1; ++i) {
////				for (int j = -1; j <= 1; ++j) {
////					int xs = curr_offset_x + i;
////					int ys = curr_offset_y + j;
////
////					int x_shift = xs, y_shift = ys; //TODO check those
////
////					int j_x, i_y, j_width, i_height;
////
////					if (y_shift < 0) { //height i
////						i_y = -y_shift;
////						i_height = tmpHeight;
////					} else {
////						i_y = 0;
////						i_height = tmpHeight - y_shift;
////					}
////
////					if (x_shift < 0) { //width j
////						j_x = -x_shift;
////						j_width = tmpWidth;
////					} else {
////						j_x = 0;
////						j_width = tmpWidth - x_shift;
////					}
////
////					dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
////					dimGrid = dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
////							((i_height) + dimBlock.y - 1) / dimBlock.y);
////
////					hipMemset(shifted_mtb[second_index * PYRAMID_LEVEL + k], 0,
////							tmpNImageSize * sizeof(uint8_t));
////					hipMemset(shifted_ebm[second_index * PYRAMID_LEVEL + k], 0,
////							tmpNImageSize * sizeof(uint8_t));
////
////					uint8_t *xor_result;
////					hipMalloc((void **) &xor_result,
////							tmpNImageSize * sizeof(uint8_t));
////
////					uint8_t *after_first_and;
////					hipMalloc((void **) &after_first_and,
////							tmpNImageSize * sizeof(uint8_t));
////
////					uint8_t *after_second_and;
////					hipMalloc((void **) &after_second_and,
////							tmpNImageSize * sizeof(uint8_t));
////
////					int* err;
////					int error;
////					hipMalloc((void **) &err, sizeof(int));
////
////					shift_Image<<<dimGrid, dimBlock, 0, stream>>>(
////							shifted_mtb[second_index * PYRAMID_LEVEL + k],
////							mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
////							tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
////					shift_Image<<<dimGrid, dimBlock, 0, stream>>>(
////							shifted_ebm[second_index * PYRAMID_LEVEL + k],
////							ebm[second_index * PYRAMID_LEVEL + k], tmpWidth,
////							tmpHeight, xs, ys, j_x, i_y, j_width, i_height);
////
////					dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
////					dimGrid = dim3(((tmpWidth) + dimBlock.x - 1) / dimBlock.x,
////							((tmpHeight) + dimBlock.y - 1) / dimBlock.y);
////
////					XOR<<<dimGrid, dimBlock, 0, stream>>>(xor_result,
////							mtb[first_index * PYRAMID_LEVEL + k],
////							shifted_mtb[second_index * PYRAMID_LEVEL + k], tmpWidth,
////							tmpNImageSize);
////
////					AND<<<dimGrid, dimBlock, 0, stream>>>(after_first_and,
////							ebm[first_index * PYRAMID_LEVEL + k], xor_result,
////							tmpWidth, tmpNImageSize);
////
////					AND<<<dimGrid, dimBlock, 0, stream>>>(after_second_and,
////							shifted_ebm[second_index * PYRAMID_LEVEL + k],
////							after_first_and, tmpWidth, tmpNImageSize);
////
////					count_Errors<<<32, 256, 0, stream>>>(after_second_and, err,
////							tmpNImageSize);
////
////					hipMemcpy(&error, err, sizeof(int), hipMemcpyDeviceToHost); //CANNOT BE ASYNC
////
////					if (error < min_error) {
////						offset_return_x = xs;
////						offset_return_y = ys;
////						min_error = error;
////					}
////					//hipFree(err);
////				}
////			}
////		}
////		cout << "found offset (x, y): " << curr_offset_x << " " << curr_offset_y
////				<< endl;
////
////	}
//
//	//hipDeviceSynchronize();
////
////	void* status;
////	for (int var = 0; var < num_threads; ++var) {
////		if (var != mid_img_index)
////			pthread_join(threads[var], &status);
////	}
//	//cout << "thread joinler bitti..." << endl;
//
//	hipDeviceSynchronize();
//
//	//cout << " ilk parttaki imajlari shiftliyoruz tek tek ..." << endl;
//
//	int eskiTotalX = 0, eskiTotalY = 0;
//	for (int m = mid_img_index - 1; m >= 0; --m) {
//
//		int x_shift = all_shifts[m].x + eskiTotalX;
//		int y_shift = all_shifts[m].y + eskiTotalY; //TODO check those
//
//		int tmpWidth = width;
//		int tmpHeight = height;
//		int tmpNImageSize = tmpWidth * tmpHeight;
//
//		hipMalloc((void**) &shifted_rgb_images[m], 3 * tmpNImageSize);
//		hipMemset(shifted_rgb_images[m], 0,
//				3 * tmpNImageSize * sizeof(uint8_t));
//
//		int j_x, i_y, j_width, i_height;
//
//		if (y_shift < 0) { //height i
//			i_y = -y_shift;
//			i_height = tmpHeight;
//		} else {
//			i_y = 0;
//			i_height = tmpHeight - y_shift;
//		}
//
//		if (x_shift < 0) { //width j
//			j_x = -x_shift;
//			j_width = tmpWidth;
//		} else {
//			j_x = 0;
//			j_width = tmpWidth - x_shift;
//		}
//
//		dim3 dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//		dim3 dimGrid = dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
//				((i_height) + dimBlock.y - 1) / dimBlock.y);
//
//		RGB_shift_Image<<<dimGrid, dimBlock, 0, streams[m]>>>(
//				shifted_rgb_images[m], d_rgb_images[m], tmpWidth, tmpHeight,
//				x_shift, y_shift, j_x, i_y, j_width, i_height);
//
//		eskiTotalX += all_shifts[m].x;
//		eskiTotalY += all_shifts[m].y;
//		//cout << "   shiftledik: x,y " << eskiTotalX << " " << eskiTotalY << endl;
//	}
//
//	eskiTotalX = 0;
//	eskiTotalY = 0;
//	for (int m = mid_img_index + 1; m < img_count; ++m) {
//
//		int x_shift = all_shifts[m].x + eskiTotalX;
//		int y_shift = all_shifts[m].y + eskiTotalY; //TODO check those
//
//		int tmpWidth = width;
//		int tmpHeight = height;
//		int tmpNImageSize = tmpWidth * tmpHeight;
//
//		hipMalloc((void**) &shifted_rgb_images[m], 3 * tmpNImageSize);
//		hipMemset(shifted_rgb_images[m], 0,
//				3 * tmpNImageSize * sizeof(uint8_t));
//
//		int j_x, i_y, j_width, i_height;
//
//		if (y_shift < 0) { //height i
//			i_y = -y_shift;
//			i_height = tmpHeight;
//		} else {
//			i_y = 0;
//			i_height = tmpHeight - y_shift;
//		}
//
//		if (x_shift < 0) { //width j
//			j_x = -x_shift;
//			j_width = tmpWidth;
//		} else {
//			j_x = 0;
//			j_width = tmpWidth - x_shift;
//		}
//
//		dim3 dimBlock = dim3(THREAD_COUNT, THREAD_COUNT);
//		dim3 dimGrid = dim3(((j_width) + dimBlock.x - 1) / dimBlock.x,
//				((i_height) + dimBlock.y - 1) / dimBlock.y);
//
//		RGB_shift_Image<<<dimGrid, dimBlock, 0, streams[m]>>>(
//				shifted_rgb_images[m], d_rgb_images[m], tmpWidth, tmpHeight,
//				x_shift, y_shift, j_x, i_y, j_width, i_height);
//
//		eskiTotalX += all_shifts[m].x;
//		eskiTotalY += all_shifts[m].y;
//		//cout << "   shiftledik: x,y " << eskiTotalX << " " << eskiTotalY << endl;
//	}
//
//	hipEventRecord (stop);
//	hipEventSynchronize(stop);
//	float milliseconds = 0;
//	hipEventElapsedTime(&milliseconds, start, stop);
//	cout << milliseconds
//			<< " ms total (including read img + memcopies, excluding write output)"
//			<< endl;
//
//	hipDeviceSynchronize();
//
//	//print original grayscale img
////	stbi_write_png(path, width, height, 3, rgb_images[mid_img_index], width*3);
//
//	tmpNImageSize = height * width;
//	tmpHeight = height;
//	tmpWidth = width;
//
//	for (int var = 0; var < img_count; ++var) {
//
//		if (var == mid_img_index) {
//			continue;
//		}
//
//		char str[12];
//		sprintf(str, "%d.png", var);
//		char path[80] = "/home/kca/Desktop/img";
//		strcat(path, str);
//
//		uint8_t* tmpmtb = (uint8_t *) malloc(
//				sizeof(uint8_t) * tmpNImageSize * 3);
//		hipMemcpy(tmpmtb, shifted_rgb_images[var],
//				sizeof(uint8_t) * tmpNImageSize * 3, hipMemcpyDeviceToHost);
//
//		stbi_write_png(path, tmpWidth, tmpHeight, 3, tmpmtb, tmpWidth * 3);
//	}
//	char str[12];
//	sprintf(str, "%d.png", mid_img_index);
//	char path[80] = "/home/kca/Desktop/img";
//	strcat(path, str);
//
//	uint8_t* tmpmtb = (uint8_t *) malloc(sizeof(uint8_t) * tmpNImageSize * 3);
//	hipMemcpy(tmpmtb, d_rgb_images[mid_img_index],
//			sizeof(uint8_t) * tmpNImageSize * 3, hipMemcpyDeviceToHost);
//
//	stbi_write_png(path, tmpWidth, tmpHeight, 3, tmpmtb, tmpWidth * 3);

	printf("Done..........\n");

	return 0;
}
